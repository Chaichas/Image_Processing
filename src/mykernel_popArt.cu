#include "hip/hip_runtime.h"
/* Question 12 */
 
#include "../inc/mykernel_popArt.h"

/*  popArt kernel ; Inspired from the original given code */
__global__ void kernel_popArt_Warhol(unsigned int* d_img, unsigned int* d_tmp, unsigned int height,unsigned int width){
   
    // Calculate the thread indices within a 2D grid
    int idx_col = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_row = threadIdx.y + blockDim.y * blockIdx.y;

    // Calculate thread index
    int idx = ((idx_row * width) + idx_col) * 3;

    // Quadrant of pixels position
    bool bottom_left = (idx_col < (width/2)) && (idx_row < (height/2));
    bool bottom_right = ((idx_row < height / 2) && (width / 2 <= idx_col) && (idx_col < width));
    bool top_left = ((height / 2 <= idx_row) && (idx_row < height) && (idx_col < width / 2));
    bool top_right = ((height / 2 <= idx_row) && (idx_row < height) && (width / 2 <= idx_col) && (idx_col < width));

    // Flip the image vertically
    if (idx_col<width && idx_row < height)
    {
        int idx_inv = ((width * height) - ((idx_row * width) + idx_col)) * 3;
        d_img[idx + 0] = d_tmp[idx_inv + 0];
        d_img[idx + 1] = d_tmp[idx_inv + 1];
        d_img[idx + 2] = d_tmp[idx_inv + 2]; 
        
    }

    // Bottom left frame
    if (bottom_left)
    {
        d_img[idx + 0] /= 2;
        d_img[idx + 1] /= 4;
        d_img[idx + 2] = 0xFF / 1.5; //bluish teint
    }

    // Bottom right frame
    if (bottom_right) 
    {
        int grey = d_img[idx + 0] * 0.299 + d_img[idx + 1] * 0.587 + d_img[idx + 2] * 0.114;
        d_img[idx + 0] = grey;
        d_img[idx + 1] = grey;
        d_img[idx + 2] = grey;
    }

    // Top left frame
    if (top_left)
    {
        d_img[idx + 0] = 0xFF / 2; //redish teint
        d_img[idx + 1] /= 2;
        d_img[idx + 2] /= 2;       
    }

    // Top right frame
    if (top_right)
    {
        d_img[idx + 0] = 0xFF - d_img[idx + 0];
        d_img[idx + 1] = 0xFF / 2;
        d_img[idx + 2] /= 4;
    }
}

/*  Run of the popArt kernel */
void run_kernel_popArt(unsigned int *d_img, unsigned int* d_tmp,  unsigned width, unsigned height, unsigned BLOCK_WIDTH) {

    // CUDA events to measure the execution time of the popArt kernel
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); //start

    // Memory allocation on GPU
    unsigned int *dk_img;
    unsigned int *dk_tmp;
    CUDA_VERIF(hipMalloc((void **)&dk_img, sizeof(unsigned int) * 3 * width * height));
    CUDA_VERIF(hipMalloc((void **)&dk_tmp, sizeof(unsigned int) * 3 * width * height));

    // Transfer data from CPU to GPU
    CUDA_VERIF(hipMemcpy(dk_img, d_img, sizeof(unsigned int) * 3 * width * height, hipMemcpyHostToDevice));
    CUDA_VERIF(hipMemcpy(dk_tmp, d_tmp, sizeof(unsigned int) * 3 * width * height, hipMemcpyHostToDevice));
  
    /*
        - Define the x-dimension of the grid
        - Take into account if the  "width" is not divided by "BLOCK_WIDTH"
    */
    int nb_block_x = width / BLOCK_WIDTH;
    if(width % BLOCK_WIDTH) nb_block_x++;

    /*
        - Define the y-dimension of the grid
        - Take into account if the  "height" is not divided by "BLOCK_WIDTH"
    */
    int nb_block_y = height / BLOCK_WIDTH;
    if(height % BLOCK_WIDTH) nb_block_y++;

    /*
        - Define the 2D grid size using dim3 structure : number of blocks
        - Define the size of each block using dim3 structure : number of threads in the block
    */
    dim3 grid_size(nb_block_x, nb_block_y);
    dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH);

    // Calling "popArt" kernel
    kernel_popArt_Warhol<<<grid_size, block_size>>>(dk_img, dk_tmp, height, width);
    
    // Transfer data back from GPU to CPU
    CUDA_VERIF(hipMemcpy(d_img, dk_img, sizeof(unsigned int) * 3 * width * height, hipMemcpyDeviceToHost));

    hipEventRecord(stop); //stop

    // Measure elapsed time
    hipEventSynchronize(stop);
    float elapsed_ms = 0;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    printf("Image of size: %dx%d\n\tExecuted with time: %f s\n", width, height, elapsed_ms/1000); //Execution time

    // Free allocated memory on GPU
    hipFree(dk_img);
    hipFree(dk_tmp);

}

/* END Question 12 */

/* Question 13 */

/*
    The implemented code divides the image into 4 quadrants, to each modifications will be performed (saturation of the color).
    While now it is only executed on the default stream, which is stream 0, each (1/4)th of the image could be executed in a different stream,
    making it a total of 4 streams.

    Default stream (s0) : (t1) : bottom-left -> (t2) : bottom-right -> (t3) : top-left -> (t4) : top-right. 
    After : (t1) : (s0) = bottom-left, (s1) = bottom-right, (s2) : top-left, (s3) : top-right

    Using 4 streams, the modifications could be executed in parallel. Thus, we can make use of the parallel potentiel of GPU and reduce
    the amount of computation time.

*/

/* END Question 13 */