#include "hip/hip_runtime.h"
/* Question 11 - c : Diapositive effect */

#include "../inc/mykernel_diapositive.h"

/*  Diapositive effect kernel on GPU */
__global__ void diapositive_effect(unsigned int *img, unsigned int width, unsigned int height){
    
    // Calculate the thread indices within a 2D grid
    int idx_col = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_line = threadIdx.y + blockDim.y * blockIdx.y;

    // Calculate the thread index
    int idx = ((idx_line * width) + idx_col) * 3;

    // Diapositive effect
    if ((idx_col < width) && (idx_line < height)) {
        
        img[idx + 0] = 255 - img[idx + 0];
        img[idx + 1] = 255 - img[idx + 1];
        img[idx + 2] = 255 - img[idx + 2];
    }

}


/*  Run of the diapositive effect kernel */
void run_diapositive_effect(unsigned int *d_img, unsigned width, unsigned height, unsigned BLOCK_WIDTH) {
    
    // CUDA events to measure the execution time of the kernel
    /*hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); */
    
    // Memory allocation on device (GPU)
    unsigned int *dk_img;
    CUDA_VERIF(hipMalloc((void **)&dk_img, sizeof(unsigned int) * 3 * width * height));
  
    // Transfer data from CPU to GPU
    CUDA_VERIF(hipMemcpy(dk_img, d_img, sizeof(unsigned int) * 3 * width * height, hipMemcpyHostToDevice));
  
    /*
        - Define the x-dimension of the grid
        - Take into account if the  "width" is not divided by "BLOCK_WIDTH"
    */
    int nb_block_x = width / BLOCK_WIDTH;
    if(width % BLOCK_WIDTH) nb_block_x++;

    /*
        - Define the y-dimension of the grid
        - Take into account if the  "height" is not divided by "BLOCK_WIDTH"
    */
    int nb_block_y = height / BLOCK_WIDTH;
    if(height % BLOCK_WIDTH) nb_block_y++;

    /*
        - Define the 2D grid size using dim3 structure : number of blocks
        - Define the size of each block using dim3 structure : number of threads in the block
    */
    dim3 grid_size(nb_block_x, nb_block_y);
    dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH);

    // Calling "diapositive_effect" kernel
    diapositive_effect<<<grid_size, block_size>>>(dk_img, width, height);
    CUDA_VERIF(hipDeviceSynchronize()); //synchronization

    // Transfer data from GPU to CPU
    CUDA_VERIF(hipMemcpy(d_img, dk_img, sizeof(unsigned int) * 3 * width * height, hipMemcpyDeviceToHost));

    /*hipEventRecord(stop); 
    hipEventSynchronize(stop);
    float elapsed_ms = 0;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    printf("Image of size: %dx%d\n\tExecuted with time: %f s\n", width, height, elapsed_ms/1000); */

    // Free allocated memory on GPU
    hipFree(dk_img);

}

/* END Question 11 - c */